#include "test_functor.h"

void CudaFree(void *p) {
  if (p == nullptr) return;
  CUDA_CHECK(hipFree(p));
}
template <typename T>
T TestMain(const std::vector<T> &cpu) {
  hipStream_t stream;
  CUDA_CHECK(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));
  CUDA_CHECK(hipStreamSynchronize(stream));

  size_t n = cpu.size();
  auto *gpu = CudaMalloc<T>(n);
  CUDA_CHECK(hipMemcpyAsync(
      gpu, cpu.data(), n * sizeof(T), hipMemcpyHostToDevice, stream));
  auto *gpu_ret = CudaMalloc<T>(1);
  auto addf = AddFunctor<float>();
  auto trans = IdentityFunctor<float, float>(n);
  hipcub::TransformInputIterator<float,
                              IdentityFunctor<float, float>,
                              const float *>
      trans_x(gpu, trans);
  size_t tmp_bytes;
  CUDA_CHECK(hipcub::DeviceReduce::Reduce(
      nullptr, tmp_bytes, trans_x, gpu_ret, n, addf, 0.0f, stream));
  std::cout << "tmp_bytes:" << tmp_bytes << std::endl;
  uint8_t *gpu_tmp = CudaMalloc<uint8_t>(tmp_bytes);
  CUDA_CHECK(hipcub::DeviceReduce::Reduce(
      gpu_tmp, tmp_bytes, trans_x, gpu_ret, n, addf, 0.0f, stream));

  T cpu_ret;
  CUDA_CHECK(hipMemcpyAsync(
      &cpu_ret, gpu_ret, sizeof(T), hipMemcpyDeviceToHost, stream));

  CUDA_CHECK(hipStreamSynchronize(stream));
  CUDA_CHECK(hipStreamDestroy(stream));

  CudaFree(gpu);
  CudaFree(gpu_ret);
  CudaFree(gpu_tmp);
  return cpu_ret;
}

float Reduce1024x100() {
  std::cout << "CUB version : " << CUB_VERSION << std::endl;
  std::vector<float> data(1024 * 100, 1);
  auto ret = TestMain(data);
  return ret;
}

void UseModuleA() {}
